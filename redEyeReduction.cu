#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "redEyeReduction.h"
#include "hip/hip_runtime.h"

#define PROFILE 1
#define BLOCK_SIZE 256
dim3 blockSize(32, 8, 1);

namespace redEyeReduction {
	void splitChannels(Mat im, unsigned char *red, unsigned char *green, unsigned char *blue) {
		size_t numPixels = im.rows*im.cols;
		unsigned char *imgPtr = new unsigned char[numPixels * im.channels()];
		unsigned char *cvPtr = im.ptr<unsigned char>(0);         // For Efficient Pixel Access
		for (size_t i = 0; i < numPixels * im.channels(); ++i) {
			imgPtr[i] = cvPtr[i];
		}
		for (size_t i = 0; i < numPixels; ++i) {
			blue[i] = imgPtr[3 * i + 0];
			green[i] = imgPtr[3 * i + 1];
			red[i] = imgPtr[3 * i + 2];
		}
		delete[] imgPtr;
	}
	
	__global__ void naive_normalized_cross_correlation(float* dev_response, unsigned char* dev_original, unsigned char* dev_template,
		int rows, int cols, int template_half_height, int template_height, int template_half_width, int template_width,
		int template_size, float template_mean)
	{
	  int2 image_index_2d = make_int2( ( blockIdx.x * blockDim.x ) + threadIdx.x, ( blockIdx.y * blockDim.y ) + threadIdx.y );
	  int  image_index_1d = (cols * image_index_2d.y) + image_index_2d.x;

	  if (image_index_2d.x < cols && image_index_2d.y < rows)
	  {
		float image_sum = 0.0f;

		for ( int y = -template_half_height; y <= template_half_height; y++ ) {
		  for ( int x = -template_half_width; x <= template_half_width; x++ )  {
			int2 image_offset_index_2d         = make_int2( image_index_2d.x + x, image_index_2d.y + y );
			int2 image_offset_index_2d_clamped = make_int2(min(cols - 1, max(0, image_offset_index_2d.x)), min(rows - 1, max(0, image_offset_index_2d.y)));
			int  image_offset_index_1d_clamped = (cols * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

			unsigned char image_offset_value = dev_original[ image_offset_index_1d_clamped ];

			image_sum += (float)image_offset_value;
		  }
		}

		float image_mean = image_sum / (float)template_size;

		float sum_of_image_template_diff_products = 0.0f;
		float sum_of_squared_image_diffs          = 0.0f;
		float sum_of_squared_template_diffs       = 0.0f;

		for ( int y = -template_half_height; y <= template_half_height; y++ ) {
		  for ( int x = -template_half_width; x <= template_half_width; x++ )  {
			int2 image_offset_index_2d         = make_int2( image_index_2d.x + x, image_index_2d.y + y );
			int2 image_offset_index_2d_clamped = make_int2(min(cols - 1, max(0, image_offset_index_2d.x)), min(rows - 1, max(0, image_offset_index_2d.y)));
			int  image_offset_index_1d_clamped = (cols * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

			unsigned char image_offset_value = dev_original[ image_offset_index_1d_clamped ];
			float         image_diff         = (float)image_offset_value - image_mean;

			int2 template_index_2d = make_int2( x + template_half_width, y + template_half_height );
			int  template_index_1d = (template_width * template_index_2d.y) + template_index_2d.x;

			unsigned char template_value = dev_template[ template_index_1d ];
			float         template_diff  = template_value - template_mean;

			float image_template_diff_product = image_offset_value   * template_diff;
			float squared_image_diff          = image_diff           * image_diff;
			float squared_template_diff       = template_diff        * template_diff;

			sum_of_image_template_diff_products += image_template_diff_product;
			sum_of_squared_image_diffs          += squared_image_diff;
			sum_of_squared_template_diffs       += squared_template_diff;
		  }
		}

		float result_value = 0.0f;

		if ( sum_of_squared_image_diffs != 0 && sum_of_squared_template_diffs != 0 ) {
		  result_value = sum_of_image_template_diff_products / sqrt( sum_of_squared_image_diffs * sum_of_squared_template_diffs );
		}
		dev_response[ image_index_1d ] = result_value;
	  }
	}

	__global__ void create_normalized_matrix(float *red, float *green, float *blue, float *combined, size_t rows, size_t cols) {
		int  ny = rows;
		int  nx = cols;
		int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
		int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

		if (image_index_2d.x < nx && image_index_2d.y < ny) {
			combined[image_index_1d] = red[image_index_1d] * green[image_index_1d] * blue[image_index_1d];
		}
	}

	__global__ void mean_shift(float* val, float constant, size_t rows, size_t cols) {
		int  ny = rows;
		int  nx = cols;
		int2 image_index_2d = make_int2((blockIdx.x * blockDim.x) + threadIdx.x, (blockIdx.y * blockDim.y) + threadIdx.y);
		int  image_index_1d = (nx * image_index_2d.y) + image_index_2d.x;

		if (image_index_2d.x < nx && image_index_2d.y < ny) {
			val[image_index_1d] = val[image_index_1d]-constant;
		}
	}

	__global__ void compute_histogram(unsigned int* dev_inputVals, unsigned int* dev_histogram, unsigned int pass, size_t numElems) {
			unsigned idx = threadIdx.x + blockIdx.x * blockDim.x;
			if (idx >= numElems) {
				return;
			}

			unsigned int bit = (dev_inputVals[idx] & (1u << pass)) >> pass;
			atomicAdd(&dev_histogram[bit], 1);
		}

	__global__ void scan_element(unsigned int* dev_inputVals, unsigned int* dev_scaned, unsigned int base, unsigned int pass,
		size_t numElems, unsigned int threadSize) {
			unsigned int idx = threadIdx.x + base * threadSize;
			if (idx >= numElems) {
				return;
			}

			unsigned int bit = (dev_inputVals[idx] & (1u << pass)) >> pass;
			dev_scaned[idx] = bit;
			__syncthreads();

			int spot, val;
			for (unsigned int s = threadSize >> 1; s > 0; s >>= 1) {
				spot = idx - s;
				if (spot >= 0 && spot >= base * threadSize) {
					val = dev_scaned[spot];
				}
				__syncthreads();
				if (spot >= 0 && spot >= base * threadSize) {
					dev_scaned[idx] += val;
				}
				__syncthreads();
			}

			if (base > 0) {
				dev_scaned[idx] += dev_scaned[base * threadSize - 1];
			}
		}

	__global__ void move_element(unsigned int* dev_inputVals, unsigned int* dev_inputPos, unsigned int* dev_outputVals, unsigned int* dev_outputPos,
		unsigned int* dev_histogram, unsigned int* dev_scaned, unsigned int pass, unsigned int numElems) {
			unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
			if (idx >= numElems) {
				return;
			}

			unsigned int bit = (dev_inputVals[idx] & (1u << pass)) >> pass;
			unsigned int offset;
			if (bit) {
				offset = dev_histogram[0];
				if (idx > 0) {
					offset += dev_scaned[idx - 1];
				}
			}
			else {
				offset = idx - dev_scaned[idx];
			}
			dev_outputVals[offset] = dev_inputVals[idx];
			dev_outputPos[offset] = dev_inputPos[idx];
		}

	void sort(unsigned int* dev_inputVals, unsigned int* dev_inputPos, unsigned int* dev_outputVals, unsigned int* dev_outputPos,
		size_t numElems)
	{
		dim3 gridSize(ceil((float)(numElems) / BLOCK_SIZE) + 1);
		dim3 blockSize(1024);

		unsigned int* dev_histogram;
		hipMalloc((void**)&dev_histogram, 2 * sizeof(unsigned int));
		unsigned int* dev_scaned;
		hipMalloc((void**)&dev_scaned, numElems * sizeof(unsigned int));

		for (unsigned int pass = 0; pass < 8 * sizeof(unsigned int); ++pass) {
			hipMemset(dev_histogram, 0, 2 * sizeof(unsigned int));
			hipMemset(dev_scaned, 0, numElems * sizeof(unsigned int));
			hipMemset(dev_outputVals, 0, numElems * sizeof(unsigned int));
			hipMemset(dev_outputPos, 0, numElems * sizeof(unsigned int));

			#if PROFILE
				GpuTimer timer;
				timer.Start();
			#endif
			compute_histogram << <gridSize, BLOCK_SIZE >> >(dev_inputVals, dev_histogram, pass, numElems);
			hipDeviceSynchronize();

			for (unsigned int base = 0; base < gridSize.x; base++) {
				scan_element << <dim3(1), 1024 >> >(dev_inputVals, dev_scaned, base, pass, numElems, blockSize.x);
				hipDeviceSynchronize();
			}

			move_element << <gridSize, BLOCK_SIZE >> >(dev_inputVals, dev_inputPos, dev_outputVals, dev_outputPos, dev_histogram, dev_scaned, pass, numElems);
			hipDeviceSynchronize();
			#if PROFILE
				timer.Stop();
				printf("sort: %f msecs.\n", timer.Elapsed());
			#endif

			hipMemcpy(dev_inputVals, dev_outputVals, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice);
			hipMemcpy(dev_inputPos, dev_outputPos, numElems * sizeof(unsigned int), hipMemcpyDeviceToDevice);
			hipDeviceSynchronize();
		}

		hipFree(dev_histogram);
		hipFree(dev_scaned);
	}

	__global__ void allocatePos(unsigned int *val, size_t size) {
		unsigned int idx = threadIdx.x + blockIdx.x * blockDim.x;
		if (idx >= size) {
			return;
		}
		val[idx] = idx;
	}

	__global__ void remove_redness_from_coordinates(unsigned int*  dev_coordinates, unsigned char* dev_blue, unsigned char* dev_green, unsigned char* dev_red_output, 
		int num_coordinates, int rows, int cols, int template_half_height, int template_half_width)
	{
		int global_index_1d = (blockIdx.x * blockDim.x) + threadIdx.x;

		int imgSize = cols * rows;

		if (global_index_1d < num_coordinates) {
			unsigned int image_index_1d = dev_coordinates[imgSize - global_index_1d - 1];
			ushort2 image_index_2d = make_ushort2(image_index_1d % cols, image_index_1d / cols);

			for (int y = image_index_2d.y - template_half_height; y <= image_index_2d.y + template_half_height; y++)
			{
				for (int x = image_index_2d.x - template_half_width; x <= image_index_2d.x + template_half_width; x++)
				{
					int2 image_offset_index_2d_clamped = make_int2(min(cols - 1, max(0, x)), min(rows - 1, max(0, y)));
					int  idx = (cols * image_offset_index_2d_clamped.y) + image_offset_index_2d_clamped.x;

					unsigned int gb_average = (dev_green[idx] + dev_blue[idx]) / 2;
					dev_red_output[idx] = (unsigned char)gb_average;
				}
			}

		}
	}
	__global__ void reduce_minmax_kernel(float* const dev_in, float* dev_out, size_t size, int minmax) {
		extern __shared__ float shared[];

		int mid = threadIdx.x + blockDim.x * blockIdx.x;
		int tid = threadIdx.x;

		if (mid < size) {
			shared[tid] = dev_in[mid];
		}
		else {
			if (minmax == 0)
				shared[tid] = FLT_MAX;
			else
				shared[tid] = -FLT_MAX;
		}

		__syncthreads();

		if (mid >= size) {
			if (tid == 0) {
				if (minmax == 0)
					dev_out[blockIdx.x] = FLT_MAX;
				else
					dev_out[blockIdx.x] = -FLT_MAX;
			}
			return;
		}

		for (unsigned int s = blockDim.x / 2; s > 0; s /= 2) {
			if (tid < s) {
				if (minmax == 0) {
					shared[tid] = min(shared[tid], shared[tid + s]);
				}
				else {
					shared[tid] = max(shared[tid], shared[tid + s]);
				}
			}

			__syncthreads();
		}

		if (tid == 0) {
			dev_out[blockIdx.x] = shared[0];
		}
	}

	float reduce_minmax(float* d_in, size_t size, int minmax) {
		size_t curr_size = size;
		float* dev_data;

		hipMalloc(&dev_data, sizeof(float)* size);
		hipMemcpy(dev_data, d_in, sizeof(float)* size, hipMemcpyDeviceToDevice);


		float* dev_temp;

		dim3 thread_dim(BLOCK_SIZE);
		const int shared_mem_size = sizeof(float)*BLOCK_SIZE;
		dim3 block_dim((int)ceil((float)size / (float)BLOCK_SIZE) + 1);
		int maxSize;
		#if PROFILE
			GpuTimer timer;
		#endif
		while (1) {
			maxSize = (int)ceil((float)curr_size / (float)BLOCK_SIZE) + 1;
			hipMalloc(&dev_temp, sizeof(float)* maxSize);

			#if PROFILE
				timer.Start();
			#endif
			reduce_minmax_kernel << <block_dim, thread_dim, shared_mem_size >> >(dev_data, dev_temp, curr_size, minmax);
			#if PROFILE
				timer.Stop();
				printf("minMax: %f msecs.\n", timer.Elapsed());
			#endif
			hipDeviceSynchronize();


			// move the current input to the output, and clear the last input if necessary
			hipFree(dev_data);
			dev_data = dev_temp;

			if (curr_size <  BLOCK_SIZE)
				break;

			curr_size = maxSize;
		}

		// theoretically we should be 
		float result;
		hipMemcpy(&result, dev_temp, sizeof(float), hipMemcpyDeviceToHost);
		hipFree(dev_temp);
		return result;
	}

	Mat gpuRER(Mat im, Mat eyeTemplate) {
		size_t rows = im.rows;
		size_t cols = im.cols;
		size_t numPixels = rows * cols;
		unsigned char *red = new  unsigned char[numPixels];
		unsigned char *green = new  unsigned char[numPixels];
		unsigned char *blue = new  unsigned char[numPixels];
		size_t rows_template = eyeTemplate.rows;
		size_t cols_template = eyeTemplate.cols;
		size_t numPixels_template = rows_template * cols_template;
		unsigned char *red_template = new  unsigned char[numPixels_template];
		unsigned char *green_template = new  unsigned char[numPixels_template];
		unsigned char *blue_template = new  unsigned char[numPixels_template];

		splitChannels(im, red, green, blue);
		splitChannels(eyeTemplate, red_template, green_template, blue_template);
		
		unsigned int r_sum = 0, g_sum = 0, b_sum = 0;
		float r_mean, g_mean, b_mean;
		for (int i = 0; i < numPixels_template; ++i)	{
			r_sum += red_template[i];
			g_sum += green_template[i];
			b_sum += blue_template[i];
		}
		r_mean = ((float)r_sum) / numPixels_template;
		g_mean = ((float)g_sum) / numPixels_template;
		b_mean = ((float)b_sum) / numPixels_template;

		unsigned char *dev_red, *dev_green, *dev_blue;
		unsigned char *dev_red_template, *dev_green_template, *dev_blue_template;
		float *dev_red_normalized, *dev_green_normalized, *dev_blue_normalized;
		float *dev_normalized;
		unsigned int *dev_input, *dev_normalized_sorted;
		unsigned int *dev_ipPosition, *dev_opPosition;
		hipMalloc((void**)&dev_red, sizeof(unsigned char)*numPixels);
		hipMalloc((void**)&dev_green, sizeof(unsigned char)*numPixels);
		hipMalloc((void**)&dev_blue, sizeof(unsigned char)*numPixels);
		hipMalloc((void**)&dev_red_template, sizeof(unsigned char)*numPixels_template);
		hipMalloc((void**)&dev_green_template, sizeof(unsigned char)*numPixels_template);
		hipMalloc((void**)&dev_blue_template, sizeof(unsigned char)*numPixels_template);
		hipMalloc((void**)&dev_red_normalized, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_green_normalized, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_blue_normalized, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_normalized, sizeof(float)*numPixels);
		hipMalloc((void**)&dev_input, sizeof(unsigned int)*numPixels);
		hipMalloc((void**)&dev_normalized_sorted, sizeof(unsigned int)*numPixels);
		hipMalloc((void**)&dev_ipPosition, sizeof(unsigned int)*numPixels);
		hipMalloc((void**)&dev_opPosition, sizeof(unsigned int)*numPixels);

		hipMemcpy(dev_red, red, sizeof(unsigned char)*numPixels, hipMemcpyHostToDevice);
		hipMemcpy(dev_green, green, sizeof(unsigned char)*numPixels, hipMemcpyHostToDevice);
		hipMemcpy(dev_blue, blue, sizeof(unsigned char)*numPixels, hipMemcpyHostToDevice);
		hipMemcpy(dev_red_template, red_template, sizeof(unsigned char)*numPixels_template, hipMemcpyHostToDevice);
		hipMemcpy(dev_green_template, green_template, sizeof(unsigned char)*numPixels_template, hipMemcpyHostToDevice);
		hipMemcpy(dev_blue_template, blue_template, sizeof(unsigned char)*numPixels_template, hipMemcpyHostToDevice);

		dim3 gridSize((cols + blockSize.x - 1) / blockSize.x, (rows + blockSize.y - 1) / blockSize.y, 1);

		#if PROFILE
			GpuTimer timer;
			timer.Start();
		#endif
		naive_normalized_cross_correlation << <gridSize, blockSize >> >(dev_red_normalized, dev_red, dev_red_template,
			rows, cols, (rows_template - 1) / 2, rows_template, (cols_template - 1) / 2, cols_template,	numPixels_template, r_mean);
		hipDeviceSynchronize();

		naive_normalized_cross_correlation << <gridSize, blockSize >> >(dev_green_normalized, dev_green, dev_green_template,
			rows, cols, (rows_template - 1) / 2, rows_template, (cols_template - 1) / 2, cols_template, numPixels_template, g_mean);
		hipDeviceSynchronize();

		naive_normalized_cross_correlation << <gridSize, blockSize >> >(dev_blue_normalized, dev_blue, dev_blue_template,
			rows, cols, (rows_template - 1) / 2, rows_template, (cols_template - 1) / 2, cols_template, numPixels_template, b_mean);
		hipDeviceSynchronize();
		#if PROFILE
			timer.Stop();
			printf("cross correlation: %f msecs.\n", timer.Elapsed());
		#endif

		#if PROFILE
			timer.Start();
		#endif
		create_normalized_matrix << < gridSize, blockSize >> >(dev_red_normalized, dev_green_normalized, dev_blue_normalized, dev_normalized,
			rows, cols);
		#if PROFILE
			timer.Stop();
			printf("normalized cross correlation: %f msecs.\n", timer.Elapsed());
		#endif

		float minVal = reduce_minmax(dev_normalized, numPixels, 0);

		#if PROFILE
			timer.Start();
		#endif
		mean_shift << < gridSize, blockSize >> >(dev_normalized, minVal, rows, cols);
		#if PROFILE
			timer.Stop();
			printf("mean shift: %f msecs.\n", timer.Elapsed());
		#endif

		dim3 gridSize2(ceil((float)(numPixels) / BLOCK_SIZE) + 1);
		allocatePos << <gridSize2, BLOCK_SIZE >> >(dev_ipPosition, numPixels);

		hipMemcpy(dev_input, dev_normalized, sizeof(unsigned int)*numPixels, hipMemcpyDeviceToDevice);
		
		sort(dev_input, dev_ipPosition, dev_normalized_sorted, dev_opPosition, numPixels);

		dim3 gridSize3((40 + BLOCK_SIZE - 1) / BLOCK_SIZE, 1, 1);
		#if PROFILE
			timer.Start();
		#endif
			remove_redness_from_coordinates << <gridSize3, BLOCK_SIZE >> >(dev_opPosition, dev_blue, dev_green,
			dev_red, 40, rows, cols, 9, 9);
		#if PROFILE
			timer.Stop();
			printf("remap: %f msecs.\n", timer.Elapsed());
		#endif
			
		hipMemcpy(red, dev_red, sizeof(unsigned char)*numPixels, hipMemcpyDeviceToHost);

		unsigned char *imgPtr = new unsigned char[numPixels * im.channels()];
		for (int i = 0; i < numPixels; ++i) {
			imgPtr[3 * i + 0] = blue[i];
			imgPtr[3 * i + 1] = green[i];
			imgPtr[3 * i + 2] = red[i];
		}

		int sizes[2];
		sizes[0] = rows;
		sizes[1] = cols;
		cv::Mat result(2, sizes, im.type(), (void *)imgPtr);

		hipFree(dev_red);
		hipFree(dev_green);
		hipFree(dev_blue);
		hipFree(dev_red_template);
		hipFree(dev_green_template);
		hipFree(dev_blue_template);
		hipFree(dev_red_normalized);
		hipFree(dev_green_normalized);
		hipFree(dev_blue_normalized);
		hipFree(dev_normalized);
		hipFree(dev_input);
		hipFree(dev_ipPosition);
		hipFree(dev_opPosition);
		hipFree(dev_normalized_sorted);

		delete[] red;
		delete[] green;
		delete[] blue;
		delete[] red_template;
		delete[] blue_template;
		delete[] green_template;

		return result;
	}
}
